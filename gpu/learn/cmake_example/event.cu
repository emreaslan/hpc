#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_one(int n, float* x, float* y) {
    int i = threadIdx.x;
    if (i < n) {
        y[i] = x[i] + 1;
    }
    // printf("thread %d finish\n", i);
}

void initialize_input(float* h_A, int n) {
    for (int i = 0; i < n; i++) {
        h_A[i] = i;
    }
}

void CUDART_CB MyCallback(hipStream_t stream, hipError_t status, void* data) {
    printf("inside callback %zu\n", (size_t)data);
}

int main(void) {
    int N = 4 * 4;
    size_t size = N * sizeof(float);

    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i) hipStreamCreate(&stream[i]);
    float* hostPtr[2];
    hipHostMalloc(&hostPtr[0], size, hipHostMallocDefault);  // allocates page-locked memory on the host
    hipHostMalloc(&hostPtr[1], size, hipHostMallocDefault);

    initialize_input(hostPtr[0], N);
    initialize_input(hostPtr[1], N);

    // Allocate vectors in device memory
    float* devPtrIn[2];
    float* devPtrOut[2];
    hipMalloc(&devPtrIn[0], size);
    hipMalloc(&devPtrIn[1], size);
    hipMalloc(&devPtrOut[0], size);
    hipMalloc(&devPtrOut[1], size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    for (size_t i = 0; i < 2; ++i) {
        hipMemcpyAsync(devPtrIn[i], hostPtr[i], size, hipMemcpyHostToDevice, stream[i]);
        add_one<<<1, N, 0, stream[i]>>>(N, devPtrIn[i], devPtrOut[i]);
        hipMemcpyAsync(hostPtr[i], devPtrOut[i], size, hipMemcpyDeviceToHost, stream[i]);
        hipStreamAddCallback(stream[i], MyCallback, (void*)i, 0);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("elapsedTime=%f\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i = 0; i < 2; ++i) hipStreamSynchronize(stream[i]);

    // Free device memory
    hipFree(&devPtrIn[0]);
    hipFree(&devPtrIn[1]);
    hipFree(&devPtrOut[0]);
    hipFree(&devPtrOut[1]);

    for (int i = 0; i < 2; ++i) hipStreamDestroy(stream[i]);

    printf("result: %f,%f\n", hostPtr[0][3], hostPtr[1][3]);

    // Free host memory
    hipHostFree(hostPtr[0]);
    hipHostFree(hostPtr[1]);
}
